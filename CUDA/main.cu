#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define IMG_PATH        "../IMG/"
#define FOREST          "forest.jpg"
#define HIKING          "hiking.jpg"
#define LOTOS           "lotos.jpg"
#define DOG_WARRIOR     "piesek_wojownik.jpg"
#define DOG             "piesek.jpg"
#define SQUIRREL        "squirrel.jpg"
#define IMAGE           IMG_PATH IMAGE_CHOICE

#define IMAGE_CHOICE    DOG

#define STB_IMAGE_IMPLEMENTATION
#include "../STB/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../STB/stb_image_write.h"

#define COLOR_MAX_VALUE             256
#define ROOT_RANK                   0

struct histogram_t{
    int red[COLOR_MAX_VALUE];
    int green[COLOR_MAX_VALUE];
    int blue[COLOR_MAX_VALUE];
};

__global__ void calculate_histogram(uint8_t *img, size_t img_size, struct histogram_t *histogram, int channels)
{
    __shared__ int local_red[COLOR_MAX_VALUE];
    __shared__ int local_green[COLOR_MAX_VALUE];
    __shared__ int local_blue[COLOR_MAX_VALUE];

    if (threadIdx.x < COLOR_MAX_VALUE)
    {
        local_red[threadIdx.x] = 0;
        local_green[threadIdx.x] = 0;
        local_blue[threadIdx.x] = 0;
    }

    int start = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = start; i< (img_size / channels); i += blockDim.x * gridDim.x)
    {
        uint8_t R = img[i * channels];
        uint8_t G = img[i * channels + 1];
        uint8_t B = img[i * channels + 2];

        atomicAdd(&local_red[R], 1);
        atomicAdd(&local_green[G], 1);
        atomicAdd(&local_blue[B], 1);
    }

    if (threadIdx.x < COLOR_MAX_VALUE)
    {
        atomicAdd(&(histogram->red[threadIdx.x]), local_red[threadIdx.x]);
        atomicAdd(&(histogram->green[threadIdx.x]), local_green[threadIdx.x]);
        atomicAdd(&(histogram->blue[threadIdx.x]), local_blue[threadIdx.x]);
    }
    
}

void save_histogram_to_csv(struct histogram_t *histogram, const char *filename) {
    FILE *file = fopen(filename, "w");
    if (!file) {
        perror("Couldn't open file to write");
        return;
    }
    fprintf(file, "Intensity,Red,Green,Blue\n");
    for (int i = 0; i < COLOR_MAX_VALUE; i++) {
        fprintf(file, "%d,%d,%d,%d\n", i, histogram->red[i], histogram->green[i], histogram->blue[i]);
    }
    fclose(file);

    printf("Saved CSV file: %s\n", filename);
}

int main(void)
{
    int width = 0;
    int height = 0;
    int channels = 0;
    float time = 0.0;
    hipEvent_t start, stop;
    size_t img_size = 0;
    unsigned char *img = NULL;
    unsigned char *gpu_img = NULL;
    struct histogram_t histogram = {0};
    struct histogram_t *gpu_histogram;
    

    img = stbi_load(IMAGE, &width, &height, &channels, 0);
    if (!img)
    {
        printf("Error in loading the image\n");
        exit(1);
    }
    printf("Loaded image with a width of %dpx, a height of %dpx and %d channels\n", 
        width, height, channels);
    img_size = width * height * channels;

    hipEventCreate(&start);
	hipEventCreate(&stop);

    hipMalloc((void **)&gpu_img, img_size * sizeof(char));
	hipMalloc((void **)&gpu_histogram, sizeof(struct histogram_t));
    hipMemcpy(gpu_img, img, img_size * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(gpu_histogram, &histogram, sizeof(struct histogram_t), hipMemcpyHostToDevice);

    hipEventRecord(start, 0);
    calculate_histogram<<<128, 512>>>(gpu_img, img_size, gpu_histogram, channels);
    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("Make histogram time is %f ms\n", time);

    hipMemcpy(&histogram, gpu_histogram, sizeof(struct histogram_t), hipMemcpyDeviceToHost);
    save_histogram_to_csv(&histogram, "histogram.csv");

    stbi_image_free(img);
	hipFree(gpu_img);
	hipFree(gpu_histogram);
}
